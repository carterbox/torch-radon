#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "cfg.h"
#include "log.h"
#include "radon.h"
#include "texture.h"
#include "utils.h"

// Assumes a launch parameters as follows: The x number of threads across the
// grid equals the x dimension of the sinogram The y number of threads across
// the grid equals the angle dimension of the sinogram the z number of threads
// across the grid equals the batch * real_channels / texture_channels
template<bool parallel_beam, int texture_channels, typename T>
__global__ void
forward_kernel(T* __restrict__ output,
               hipTextureObject_t texture,
               const float* __restrict__ angles,
               const VolumeCfg vol_cfg,
               const ProjectionCfg proj_cfg,
               const int angle_batch_size,
               const int real_channels)
{

  // Calculate sinogram coordinates
  const int ray_id = blockIdx.x * blockDim.x + threadIdx.x;
  const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
  // batch_id is actually combination of batches and real_channels
  const int batch_id = blockIdx.z * blockDim.z + threadIdx.z;
  const int angle_offset =
    proj_cfg.n_angles *
    ((batch_id / real_channels * texture_channels) % angle_batch_size);

  // Assuming the output dimensions are (batch, channel, angle, ray)
  // base is the the memory location which starts a multi-channel sinograms
  const int base =
    ray_id + proj_cfg.det_count_u *
               (angle_id + proj_cfg.n_angles * (batch_id * texture_channels));
  // mem_pitch is the memory stride between channels
  const int mem_pitch = proj_cfg.det_count_u * proj_cfg.n_angles;

  if (angle_id < proj_cfg.n_angles && ray_id < proj_cfg.det_count_u) {
    float accumulator[texture_channels];

#pragma unroll
    for (int i = 0; i < texture_channels; i++)
      accumulator[i] = 0.0f;

    // compute ray
    float sx, sy, ex, ey;
    if (parallel_beam) {
      sx =
        (ray_id - proj_cfg.det_count_u * 0.5f + 0.5f) * proj_cfg.det_spacing_u;
      sy = hypot(abs(vol_cfg.dx) + vol_cfg.width * 0.5f,
                 abs(vol_cfg.dy) + vol_cfg.height * 0.5f);
      ex = sx;
      ey = -sy;
    } else {
      sy = proj_cfg.s_dist;
      sx = 0.0f;
      ey = -proj_cfg.d_dist;
      ex =
        (ray_id - proj_cfg.det_count_u * 0.5f + 0.5f) * proj_cfg.det_spacing_u;
    }

    // rotate ray
    const float angle = angles[angle_id + angle_offset];
    const float cs = __cosf(angle);
    const float sn = __sinf(angle);

    // start position rs and direction rd (in detector coordinate system)
    float rsx = sx * cs + sy * sn;
    float rsy = -sx * sn + sy * cs;
    float rdx = ex * cs + ey * sn - rsx;
    float rdy = -ex * sn + ey * cs - rsy;

    // convert coordinates to volume coordinate system
    const float vol_orig_x = vol_cfg.dx - 0.5f * vol_cfg.width * vol_cfg.sx;
    const float vol_orig_y = vol_cfg.dy - 0.5f * vol_cfg.height * vol_cfg.sy;
    rsx = (rsx - vol_orig_x) * vol_cfg.inv_scale_x;
    rsy = (rsy - vol_orig_y) * vol_cfg.inv_scale_y;
    rdx = rdx * vol_cfg.inv_scale_x;
    rdy = rdy * vol_cfg.inv_scale_y;

    // clip to volume (to reduce memory reads)
    float dx = rdx >= 0 ? max(rdx, 1e-6) : min(rdx, -1e-6);
    float dy = rdy >= 0 ? max(rdy, 1e-6) : min(rdy, -1e-6);

    const float alpha_x_m = (-rsx) / dx;
    const float alpha_x_p = (vol_cfg.width - rsx) / dx;
    const float alpha_y_m = (-rsy) / dy;
    const float alpha_y_p = (vol_cfg.height - rsy) / dy;
    const float alpha_s =
      max(min(alpha_x_p, alpha_x_m), min(alpha_y_p, alpha_y_m));
    const float alpha_e =
      min(max(alpha_x_p, alpha_x_m), max(alpha_y_p, alpha_y_m));

    // if ray volume intersection is empty exit
    if (alpha_s > alpha_e - 1e-6) {
#pragma unroll
      for (int b = 0; b < texture_channels; b++)
        output[base + b * mem_pitch] = static_cast<T>(0.0);
      return;
    }

    rsx += rdx * alpha_s;
    rsy += rdy * alpha_s;
    rdx *= (alpha_e - alpha_s);
    rdy *= (alpha_e - alpha_s);

    const int n_steps = __float2int_rn(max(abs(rdx), abs(rdy)));
    const float vx = rdx / max(abs(rdx), abs(rdy));
    const float vy = rdy / max(abs(rdx), abs(rdy));
    const float n = hypot(vx * vol_cfg.sx, vy * vol_cfg.sy);

    float step;
    if (abs(rdy) >= abs(rdx)) {
      float y_increment = 0.5f - rsy + __float2int_rn(rsy);
      step = y_increment / vy;
      step += vy < 0;
    } else {
      float x_increment = 0.5f - rsx + __float2int_rn(rsx);
      step = x_increment / vx;
      step += vx < 0;
    }
    rsx += step * vx;
    rsy += step * vy;

#pragma unroll(4)
    for (int j = 0; j < n_steps; j++) {
      if (texture_channels == 1) {
        accumulator[0] += tex2DLayered<float>(texture, rsx, rsy, batch_id);
      } else {
        float4 read = tex2DLayered<float4>(texture, rsx, rsy, batch_id);
        accumulator[0] += read.x;
        accumulator[1] += read.y;
        accumulator[2] += read.z;
        accumulator[3] += read.w;
      }
      rsx += vx;
      rsy += vy;
    }

#pragma unroll
    for (int b = 0; b < texture_channels; b++)
      output[base + b * mem_pitch] = accumulator[b] * n;
  }
}

template<typename T>
void
radon::forward_cuda(const T* x,
                    const float* angles,
                    T* y,
                    TextureCache& tex_cache,
                    const VolumeCfg& vol_cfg,
                    const ProjectionCfg& proj_cfg,
                    const ExecCfg& exec_cfg,
                    const int batch_size,
                    const int channels,
                    const int device,
                    const int angle_batch_size)
{
  constexpr bool is_float = std::is_same<T, float>::value;
  constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;

  LOG_DEBUG("Radon forward 2D. Height: " << vol_cfg.height
                                         << " width: " << vol_cfg.width
                                         << " channels: " << channels);
  LOG_DEBUG("Radon forward 2D. Det count: "
            << proj_cfg.det_count_u << " angles: " << proj_cfg.n_angles
            << " angles_batch_size: " << angle_batch_size
            << " batch_size: " << batch_size);

  // If the number of channels is a multiple of 4, then we can use the texture
  // channels to decrease the thread grid size. NOTE: CUDA also supports
  // textures with 2 channels.
  int texture_channels = 1;
  if (channels % 4 == 0) {
    texture_channels = 4;
  }
  const int grid_size_z = batch_size * channels / texture_channels;

  // copy x into CUDA Array (allocating it if needed) and bind to texture
  Texture* tex = tex_cache.get({ device,
                                 grid_size_z,
                                 vol_cfg.height,
                                 vol_cfg.width,
                                 true,
                                 texture_channels,
                                 precision });
  tex->put(x);

  // Invoke kernel
  const dim3 grid_dim = exec_cfg.get_grid_size(
    proj_cfg.det_count_u, proj_cfg.n_angles, grid_size_z);
  const dim3 block_dim = exec_cfg.get_block_dim();

  LOG_DEBUG("Block Size x:" << block_dim.x << " y:" << block_dim.y
                            << " z:" << block_dim.z);
  LOG_DEBUG("Grid Size x:" << grid_dim.x << " y:" << grid_dim.y
                           << " z:" << grid_dim.z);

  switch (channels) {
    case 1:
      if (proj_cfg.projection_type == FANBEAM) {
        forward_kernel<false, 1, T><<<grid_dim, block_dim>>>(y,
                                                             tex->texture,
                                                             angles,
                                                             vol_cfg,
                                                             proj_cfg,
                                                             angle_batch_size,
                                                             channels);
      } else {
        forward_kernel<true, 1, T><<<grid_dim, block_dim>>>(y,
                                                            tex->texture,
                                                            angles,
                                                            vol_cfg,
                                                            proj_cfg,
                                                            angle_batch_size,
                                                            channels);
      }
      break;
    case 4:
      if (proj_cfg.projection_type == FANBEAM) {
        forward_kernel<false, 4, T><<<grid_dim, block_dim>>>(y,
                                                             tex->texture,
                                                             angles,
                                                             vol_cfg,
                                                             proj_cfg,
                                                             angle_batch_size,
                                                             channels);
      } else {
        forward_kernel<true, 4, T><<<grid_dim, block_dim>>>(y,
                                                            tex->texture,
                                                            angles,
                                                            vol_cfg,
                                                            proj_cfg,
                                                            angle_batch_size,
                                                            channels);
      }
      break;
    default:
      throw std::invalid_argument("This is an unsupported number of channels!");
  }
}

template void
radon::forward_cuda<float>(const float* x,
                           const float* angles,
                           float* y,
                           TextureCache& tex_cache,
                           const VolumeCfg& vol_cfg,
                           const ProjectionCfg& proj_cfg,
                           const ExecCfg& exec_cfg,
                           const int batch_size,
                           const int channels,
                           const int device,
                           const int angle_batch_size);

template void
radon::forward_cuda<__half>(const __half* x,
                            const float* angles,
                            __half* y,
                            TextureCache& tex_cache,
                            const VolumeCfg& vol_cfg,
                            const ProjectionCfg& proj_cfg,
                            const ExecCfg& exec_cfg,
                            const int batch_size,
                            const int channels,
                            const int device,
                            const int angle_batch_size);

// Assumes a launch parameters as follows
// The x number of threads across the grid equals the u dimension of the
// sinogram The y number of threads across the grid equals the angle dimension
// of the sinogram the z number of threads across the grid equals the v
// dimension of the sinogram
template<int channels, typename T>
__global__ void
forward_kernel_3d(T* __restrict__ output,
                  hipTextureObject_t texture,
                  const float* __restrict__ angles,
                  const VolumeCfg vol_cfg,
                  const ProjectionCfg proj_cfg)
{
  // Calculate sensor coordinates in pixels
  // TODO is there an "optimal" map from thread to coordinates that maximizes
  // cache hits?
  // TODO check other permutations (combined with different block sizes)
  const int pu = blockIdx.x * blockDim.x + threadIdx.x;
  const int angle_id = blockIdx.y * blockDim.y + threadIdx.y;
  const int pv = blockIdx.z * blockDim.z + threadIdx.z;

  // Assuming the output dimensions are (angle, v, u)
  // index is the the memory location which stats a multi-channel sinograms
  const uint index =
    (angle_id * proj_cfg.det_count_v + pv) * proj_cfg.det_count_u + pu;
  // mem_pitch is the memory stride between channels
  const uint mem_pitch =
    proj_cfg.n_angles * proj_cfg.det_count_v * proj_cfg.det_count_u;

  if (angle_id < proj_cfg.n_angles && pu < proj_cfg.det_count_u &&
      pv < proj_cfg.det_count_v) {
    // define accumulator
    float accumulator[channels];
#pragma unroll
    for (int i = 0; i < channels; i++)
      accumulator[i] = 0.0f;

    // compute ray
    const float angle = angles[angle_id];
    const float cs = __cosf(angle);
    const float sn = __sinf(angle);

    float sx = 0.0f;
    float sy = -proj_cfg.s_dist;
    // sz = initial_z + pitch * angle / (2*pi);
    float rsz = proj_cfg.initial_z + proj_cfg.pitch * angle * 0.1591549f;

    float ex =
      (pu - proj_cfg.det_count_u * 0.5f + 0.5f) * proj_cfg.det_spacing_u;
    float ey = proj_cfg.d_dist;
    // z is not affected by rotation
    float rdz =
      (pv - proj_cfg.det_count_v * 0.5f + 0.5f) * proj_cfg.det_spacing_v;

    // rotate start position rs and direction rd
    float rsx = sx * cs - sy * sn;
    float rsy = sx * sn + sy * cs;
    float rdx = ex * cs - ey * sn - rsx;
    float rdy = ex * sn + ey * cs - rsy;

    // convert coordinates to volume coordinate system
    const float vol_orig_x = vol_cfg.dx - 0.5f * vol_cfg.width * vol_cfg.sx;
    const float vol_orig_y = vol_cfg.dy - 0.5f * vol_cfg.height * vol_cfg.sy;
    const float vol_orig_z = vol_cfg.dz - 0.5f * vol_cfg.depth * vol_cfg.sz;
    rsx = (rsx - vol_orig_x) * vol_cfg.inv_scale_x;
    rsy = (rsy - vol_orig_y) * vol_cfg.inv_scale_y;
    rsz = (rsz - vol_orig_z) * vol_cfg.inv_scale_z;
    rdx = rdx * vol_cfg.inv_scale_x;
    rdy = rdy * vol_cfg.inv_scale_y;
    rdz = rdz * vol_cfg.inv_scale_z;

    // Clip ray to cube to reduce the number of memory reads
    float dx = rdx >= 0 ? max(rdx, 1e-6) : min(rdx, -1e-6);
    float dy = rdy >= 0 ? max(rdy, 1e-6) : min(rdy, -1e-6);
    float dz = rdz >= 0 ? max(rdz, 1e-6) : min(rdz, -1e-6);

    const float alpha_x_m = (-rsx) / dx;
    const float alpha_x_p = (vol_cfg.width - rsx) / dx;
    const float alpha_y_m = (-rsy) / dy;
    const float alpha_y_p = (vol_cfg.height - rsy) / dy;
    const float alpha_z_m = (-rsz) / dz;
    const float alpha_z_p = (vol_cfg.depth - rsz) / dz;

    const float alpha_s =
      max(min(alpha_x_p, alpha_x_m),
          max(min(alpha_y_p, alpha_y_m), min(alpha_z_p, alpha_z_m)));
    const float alpha_e =
      min(max(alpha_x_p, alpha_x_m),
          min(max(alpha_y_p, alpha_y_m), max(alpha_z_p, alpha_z_m)));

    if (alpha_s > alpha_e - 1e-6) {
#pragma unroll
      for (int b = 0; b < channels; b++)
        output[b * mem_pitch + index] = static_cast<T>(0.0);
      return;
    }

    rsx += rdx * alpha_s;
    rsy += rdy * alpha_s;
    rsz += rdz * alpha_s;
    rdx *= (alpha_e - alpha_s);
    rdy *= (alpha_e - alpha_s);
    rdz *= (alpha_e - alpha_s);

    // accumulate loop
    const float f_n_steps = max(abs(rdx), max(abs(rdy), abs(rdz)));
    const int n_steps = __float2uint_ru(f_n_steps);
    const float vx = rdx / f_n_steps;
    const float vy = rdy / f_n_steps;
    const float vz = rdz / f_n_steps;
    const float n = norm3df(vx * vol_cfg.sx, vy * vol_cfg.sy, vz * vol_cfg.sz);

    float step;
    if (abs(rdy) >= abs(rdx)) {
      float y_increment = 0.5f - rsy + __float2int_rn(rsy);
      step = y_increment / vy;
      step += vy < 0;
    } else {
      float x_increment = 0.5f - rsx + __float2int_rn(rsx);
      step = x_increment / vx;
      step += vx < 0;
    }
    rsx += step * vx;
    rsy += step * vy;
    rsz += step * vz;

#pragma unroll(4)
    for (int j = 0; j <= n_steps; j++) {
      if (channels == 1) {
        accumulator[0] += tex3D<float>(texture, rsx, rsy, rsz);
      } else {
        float4 read = tex3D<float4>(texture, rsx, rsy, rsz);
        accumulator[0] += read.x;
        accumulator[1] += read.y;
        accumulator[2] += read.z;
        accumulator[3] += read.w;
      }
      rsx += vx;
      rsy += vy;
      rsz += vz;
    }

    // output
#pragma unroll
    for (int b = 0; b < channels; b++) {
      output[b * mem_pitch + index] = accumulator[b] * n;
    }
  }
}

template<typename T>
void
radon::forward_cuda_3d(const T* x,
                       const float* angles,
                       T* y,
                       TextureCache& tex_cache,
                       const VolumeCfg& vol_cfg,
                       const ProjectionCfg& proj_cfg,
                       const ExecCfg& exec_cfg,
                       const int batch_size,
                       const int channels,
                       const int device,
                       const int angle_batch_size)
{
  constexpr bool is_float = std::is_same<T, float>::value;
  constexpr int precision = is_float ? PRECISION_FLOAT : PRECISION_HALF;

  // If the number of channels is a multiple of 4, then we can use the texture
  // channels to decrease the thread grid size. NOTE: CUDA also supports
  // textures with 2 channels.
  int texture_channels = 1;
  if (channels % 4 == 0) {
    texture_channels = 4;
  }
  const int textures_per_batch = channels / texture_channels;

  Texture* tex = tex_cache.get({ device,
                                 vol_cfg.depth,
                                 vol_cfg.height,
                                 vol_cfg.width,
                                 false,
                                 texture_channels,
                                 precision });

  const dim3 grid_dim = exec_cfg.get_grid_size(
    proj_cfg.det_count_u, proj_cfg.n_angles, proj_cfg.det_count_v);
  const dim3 block_dim = exec_cfg.get_block_dim();

  for (int kernel_start = 0; kernel_start < batch_size * channels;
       kernel_start += texture_channels) {
    T* local_y = &y[kernel_start * (proj_cfg.det_count_u *
                                    proj_cfg.det_count_v * proj_cfg.n_angles)];
    tex->put(
      &x[kernel_start * (vol_cfg.depth * vol_cfg.height * vol_cfg.width)]);
    const float* langles =
      &angles[((kernel_start / channels) % angle_batch_size) *
              proj_cfg.n_angles];

    // Invoke kernel
    if (texture_channels == 1) {
      forward_kernel_3d<1><<<grid_dim, block_dim>>>(
        local_y, tex->texture, langles, vol_cfg, proj_cfg);
    } else {
      forward_kernel_3d<4><<<grid_dim, block_dim>>>(
        local_y, tex->texture, langles, vol_cfg, proj_cfg);
    }
  }
}

template void
radon::forward_cuda_3d<float>(const float* x,
                              const float* angles,
                              float* y,
                              TextureCache& tex_cache,
                              const VolumeCfg& vol_cfg,
                              const ProjectionCfg& proj_cfg,
                              const ExecCfg& exec_cfg,
                              const int batch_size,
                              const int channels,
                              const int device,
                              const int angle_batch_size);

template void
radon::forward_cuda_3d<__half>(const __half* x,
                               const float* angles,
                               __half* y,
                               TextureCache& tex_cache,
                               const VolumeCfg& vol_cfg,
                               const ProjectionCfg& proj_cfg,
                               const ExecCfg& exec_cfg,
                               const int batch_size,
                               const int channels,
                               const int device,
                               const int angle_batch_size);
