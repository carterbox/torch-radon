#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <iostream>
#include <string>

#include "defines.h"
#include "texture.h"
#include "utils.h"

TextureConfig::TextureConfig(int device,
                             int depth,
                             int height,
                             int width,
                             bool layered,
                             int channels,
                             int precision)
  : device(device)
  , depth(depth)
  , height(height)
  , width(width)
  , is_layered(layered)
  , channels(channels)
  , precision(precision)
{
}

bool
TextureConfig::operator==(const TextureConfig& o) const
{
  return this->device == o.device && this->width == o.width &&
         this->height == o.height && this->is_layered == o.is_layered &&
         this->depth == o.depth && this->channels == o.channels &&
         this->precision == o.precision;
}

TextureType
TextureConfig::get_texture_type() const
{
  if (this->is_layered && this->height == 0)
    return TEX_1D_LAYERED;
  if (this->is_layered)
    return TEX_2D_LAYERED;
  return TEX_3D;
}

TextureConfig
create_1Dlayered_texture_config(int device,
                                int size,
                                int layers,
                                int channels,
                                int precision)
{
  return TextureConfig(device, layers, 0, size, true, channels, precision);
}

std::ostream&
operator<<(std::ostream& os, TextureConfig const& m)
{
  std::string precision = m.precision == PRECISION_FLOAT ? "float" : "half";

  return os << "(device: " << m.device << ", depth: " << m.depth
            << ", height: " << m.height << ", width: " << m.width
            << ", channels: " << m.channels << ", precision: " << precision
            << ", " << (m.is_layered ? "layered" : "not layered") << ")";
}

/// Assume data ordered        (height, channel, width) for 1D layered
/// Assume data ordered (depth, channel, height, width) for 2D layered
/// Assume data ordered (channel, depth, height, width) for 3D
template<int texture_type, typename T>
__global__ void
write_to_surface(const float* data,
                 hipSurfaceObject_t surface,
                 const int width,
                 const int height,
                 const int depth)
{
  constexpr int channels = sizeof(T) / 4;
  static_assert(std::is_same<T, float1>::value ||
                  std::is_same<T, float2>::value ||
                  std::is_same<T, float4>::value,
                "Only float1, float2, and float4 are supported.");

  int pitch;
  switch (texture_type) {
    case TEX_1D_LAYERED:
      pitch = width;
      break;
    case TEX_2D_LAYERED:
      pitch = height * width;
      break;
    case TEX_3D:
      pitch = depth * height * width;
      break;
  }

  for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < width;
       x += blockDim.x * gridDim.x) {
    for (int y = blockIdx.y * blockDim.y + threadIdx.y; y < height;
         y += blockDim.y * gridDim.y) {
      for (int z = blockIdx.z * blockDim.z + threadIdx.z; z < depth;
           z += blockDim.z * gridDim.z) {

        int offset;
        switch (texture_type) {
          case TEX_1D_LAYERED:
            offset = ((y)*channels + 0) * width + x;
            break;
          case TEX_2D_LAYERED:
            offset = (((z)*channels + 0) * height + y) * width + x;
            break;
          case TEX_3D:
            offset = (((0) * depth + z) * height + y) * width + x;
            break;
        }

        T tmp;
        if constexpr (channels >= 1) {
          tmp.x = data[0 * pitch + offset];
        }
        if constexpr (channels >= 2) {
          tmp.y = data[1 * pitch + offset];
        }
        if constexpr (channels == 4) {
          tmp.z = data[2 * pitch + offset];
          tmp.w = data[3 * pitch + offset];
        }

        switch (texture_type) {
          case TEX_1D_LAYERED:
            surf1DLayeredwrite<T>(tmp, surface, x * sizeof(T), y);
            break;
          case TEX_2D_LAYERED:
            surf2DLayeredwrite<T>(tmp, surface, x * sizeof(T), y, z);
            break;
          case TEX_3D:
            surf3Dwrite<T>(tmp, surface, x * sizeof(T), y, z);
            break;
        }
      }
    }
  }
}

template<int texture_type>
__global__ void
write_half_to_surface(const __half* data,
                      hipSurfaceObject_t surface,
                      const int width,
                      const int height,
                      const int depth)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  const int z = blockIdx.z * blockDim.z + threadIdx.z;

  if (x < width && y < height && z < depth) {
    const int pitch = width * height * depth;
    const int offset = (z * height + y) * width + x;

    __half tmp[4];
    for (int i = 0; i < 4; i++)
      tmp[i] = data[i * pitch + offset];

    switch (texture_type) {
      case TEX_1D_LAYERED:
        surf1DLayeredwrite<float2>(
          *(float2*)tmp, surface, x * sizeof(float2), y);
        break;
      case TEX_2D_LAYERED:
        surf2DLayeredwrite<float2>(
          *(float2*)tmp, surface, x * sizeof(float2), y, z);
        break;
      case TEX_3D:
        surf3Dwrite<float2>(*(float2*)tmp, surface, x * sizeof(float2), y, z);
        break;
    }
  }
}

hipChannelFormatDesc
get_channel_desc(int channels, int precision)
{
  if (precision == PRECISION_FLOAT) {
    if (channels == 1) {
      return hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    }
    if (channels == 4) {
      return hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    }
  }
  if (precision == PRECISION_HALF && channels == 4) {
    return hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindFloat);
  }

  LOG_WARNING("Unsupported number of channels and precision (channels:"
              << channels << ", precision: " << precision << ")");
  return hipCreateChannelDesc(16, 16, 16, 16, hipChannelFormatKindFloat);
}

Texture::Texture(TextureConfig c)
  : cfg(c)
{
  checkCudaErrors(hipSetDevice(this->cfg.device));

  LOG_INFO("Allocating Texture " << this->cfg);

  // Allocate CUDA array
  hipChannelFormatDesc channelDesc =
    get_channel_desc(cfg.channels, cfg.precision);
  auto allocation_type = cfg.is_layered ? hipArrayLayered : hipArrayDefault;

  const hipExtent extent = make_hipExtent(cfg.width, cfg.height, cfg.depth);
  checkCudaErrors(
    hipMalloc3DArray(&array, &channelDesc, extent, allocation_type));

  // Create resource descriptor
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = array;

  // Specify texture object parameters
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeBorder;
  texDesc.addressMode[1] = hipAddressModeBorder;
  texDesc.addressMode[2] = hipAddressModeBorder;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  // Create texture object
  checkCudaErrors(hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL));

  // Create surface object
  checkCudaErrors(hipCreateSurfaceObject(&surface, &resDesc));
}

void
Texture::put(const float* data)
{
  if (this->cfg.precision == PRECISION_HALF) {
    LOG_WARNING("Putting half precision data into a float texture");
  }

  checkCudaErrors(hipSetDevice(this->cfg.device));

  // Pytorch (channel first) has a different memory order than CUDA textures
  // (channel last), so we have to use a special copy method when using
  // multiple channels
  if (cfg.channels == 1) {
    // if using a single channel use hipMemcpy to copy data into array
    hipMemcpy3DParms myparms = { 0 };
    myparms.srcPos = make_hipPos(0, 0, 0);
    myparms.dstPos = make_hipPos(0, 0, 0);
    myparms.srcPtr = make_hipPitchedPtr(
      (void*)data, cfg.width * sizeof(float), cfg.width, max(cfg.height, 1));
    myparms.dstArray = this->array;

    myparms.extent = make_hipExtent(cfg.width, max(cfg.height, 1), cfg.depth);

    myparms.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&myparms));
  } else if (cfg.channels == 4) {
    // else if using multiple channels use custom kernel to copy the data
    int texture_type = cfg.get_texture_type();
    if (texture_type == TEX_1D_LAYERED) {
      dim3 grid_dim(roundup_div(cfg.width, 16), roundup_div(cfg.depth, 16));
      LOG_DEBUG("[TORCH RADON] Copying 1D Texture " << this->cfg);
      write_to_surface<TEX_1D_LAYERED, float4>
        <<<grid_dim, dim3(16, 16)>>>(data,
                                     this->surface,
                                     max(cfg.width, 1),
                                     max(cfg.height, 1),
                                     max(cfg.depth, 1));
    } else {
      dim3 grid_dim(
        roundup_div(cfg.width, 16), roundup_div(cfg.height, 16), cfg.depth);
      if (texture_type == TEX_2D_LAYERED) {
        LOG_DEBUG("[TORCH RADON] Copying 2D Texture " << this->cfg);
        write_to_surface<TEX_2D_LAYERED, float4>
          <<<dim3(1, 1, 1), dim3(1, 1, 1)>>>(data,
                                             this->surface,
                                             max(cfg.width, 1),
                                             max(cfg.height, 1),
                                             max(cfg.depth, 1));
      } else {
        LOG_DEBUG("[TORCH RADON] Copying 3D Texture " << this->cfg);
        write_to_surface<TEX_3D, float4>
          <<<grid_dim, dim3(16, 16)>>>(data,
                                       this->surface,
                                       max(cfg.width, 1),
                                       max(cfg.height, 1),
                                       max(cfg.depth, 1));
      }
    }
#ifdef DEBUG
    checkCudaErrors(hipDeviceSynchronize());
#endif
  } else {
    throw std::invalid_argument("There can only be 1 or 4 texture channels!");
  }
}

void
Texture::put(const __half* data)
{
  if (this->cfg.precision == PRECISION_FLOAT) {
    LOG_WARNING("Putting single precision data into a half precision texture");
  }

  checkCudaErrors(hipSetDevice(this->cfg.device));

  int texture_type = cfg.get_texture_type();
  if (texture_type == TEX_1D_LAYERED) {
    dim3 grid_dim(roundup_div(cfg.width, 16), roundup_div(cfg.depth, 16));
    write_half_to_surface<TEX_1D_LAYERED><<<grid_dim, dim3(16, 16)>>>(
      (__half*)data, this->surface, cfg.width, cfg.depth, 1);
  } else {
    dim3 grid_dim(
      roundup_div(cfg.width, 16), roundup_div(cfg.height, 16), cfg.depth);
    if (texture_type == TEX_2D_LAYERED) {
      write_half_to_surface<TEX_2D_LAYERED><<<grid_dim, dim3(16, 16)>>>(
        (__half*)data, this->surface, cfg.width, cfg.height, cfg.depth);
    } else {
      write_half_to_surface<TEX_3D><<<grid_dim, dim3(16, 16)>>>(
        (__half*)data, this->surface, cfg.width, cfg.height, cfg.depth);
    }
  }
}

bool
Texture::matches(TextureConfig& c)
{
  return c == this->cfg;
}

Texture::~Texture()
{
  LOG_DEBUG("[TORCH RADON] Freeing Texture " << this->cfg);

  if (this->array != nullptr) {
    checkCudaErrors(hipSetDevice(this->cfg.device));
    checkCudaErrors(hipDestroyTextureObject(this->texture));
    checkCudaErrors(hipDestroySurfaceObject(this->surface));
    checkCudaErrors(hipFreeArray(this->array));
    this->array = nullptr;
  }
}
